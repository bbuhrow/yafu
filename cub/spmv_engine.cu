#include <stdio.h>
#include <vector>
#include <device_unaryspmv.cuh>

#include "spmv_engine.h"

#if defined(_WIN32) || defined (_WIN64)
	#define SPMV_ENGINE_DECL __declspec(dllexport)
#else
	#define SPMV_ENGINE_DECL __attribute__((visibility("default")))
#endif

#define CUDA_TRY(func) \
        {                                                               \
                hipError_t status = func;                              \
                if (status != (hipError_t) hipSuccess) {             \
                        const char * str = hipGetErrorString(status);  \
                        if (!str)                                       \
                                str = "Unknown";                        \
                        printf("error (%s:%d): %s\n", __FILE__, __LINE__, str);\
                        exit(-1);                                       \
                }                                                       \
        }

typedef unsigned int uint32;

struct spmv_engine
{
        spmv_engine()
          : temp_data(0), temp_size(0)
        {
        }

        ~spmv_engine()
        {
                if (temp_size)
                        CUDA_TRY(hipFree(temp_data))
        }

        void * temp_data;
        size_t temp_size;
};

__device__ v_t operator+(const v_t& left, const v_t& right) {
	return v_xor(left, right);
};

extern "C"
{

SPMV_ENGINE_DECL void * 
spmv_engine_init(int * vbits)
{
	*vbits = VBITS;
	return new spmv_engine;	
}

SPMV_ENGINE_DECL void 
spmv_engine_free(void *e)
{
	delete (spmv_engine *)e;
}

SPMV_ENGINE_DECL void 
spmv_engine_run(void * e, spmv_data_t * data)
{
	spmv_engine *engine = (spmv_engine *)e;
	size_t temp_size;

	DeviceUnarySpmv::CsrMV(NULL, temp_size, 
		(int *)data->row_entries, (int *)data->col_entries, (v_t *)data->vector_in, (v_t *)data->vector_out,
		data->num_rows, data->num_cols, data->num_col_entries, v_zero);

	if (temp_size > engine->temp_size) {
		if (engine->temp_size) CUDA_TRY(hipFree(engine->temp_data))
		CUDA_TRY(hipMalloc(&engine->temp_data, temp_size))
		engine->temp_size = temp_size;
		printf("Allocated %0.1f MB for SpMV library\n", (double)temp_size / 1048576);
	}

	// Run SpMV: y = A x + y
	DeviceUnarySpmv::CsrMV(engine->temp_data, temp_size,
		(int *)data->row_entries, (int *)data->col_entries, (v_t *)data->vector_in, (v_t *)data->vector_out,
		data->num_rows, data->num_cols, data->num_col_entries, v_zero);
}

} // extern "C"

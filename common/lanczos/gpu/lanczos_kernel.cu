#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "lanczos_gpu_core.h"

#ifdef __cplusplus
extern "C" {
#endif

/*------------------------------------------------------------------------*/
__global__ void
lanczos_kernel_mask(v_t *x, v_t mask, uint32 n)
{
	uint32 i;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;

	for (i = grid_id; i < n; i += num_threads)
		x[i] = v_and(x[i], mask);
}

/*------------------------------------------------------------------------*/
__global__ void
lanczos_kernel_xor(v_t *dest, v_t *src, uint32 n)
{
	uint32 i;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;

	for (i = grid_id; i < n; i += num_threads)
		dest[i] = v_xor(dest[i], src[i]);
}

/*------------------------------------------------------------------------*/
__global__ void
lanczos_kernel_inner_prod(v_t *y, v_t *v,
			v_t *x, uint32 n)
{
	uint32 i, j;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;
	v_t acc;
	__shared__ v_t c[32*VWORDS][3];

	for (i = threadIdx.x; i < 32 * VWORDS; i += blockDim.x) {
		acc = x[2 * i];
		c[i][0] = acc;

		acc = v_xor(acc, x[2 * i + 1]);
		c[i][2] = acc;

		acc = v_xor(acc, x[2 * i]);
		c[i][1] = acc;
	}

	__syncthreads();

	for (i = grid_id; i < n; i += num_threads) {
		v_t vi = v[i];
		for (j = 0; j < VWORDS; j++) acc.w[j] = 0;

		for (j = 0; j < 32 * VWORDS; j++) {
			uint32 k = (vi.w[j >> 5] >> (2*(j & 31))) & 3;
			if (k != 0) acc = v_xor(acc, c[j][k-1]);
		}
		y[i] = v_xor(y[i], acc);
	}
}

/*------------------------------------------------------------------------*/

/* thanks to Patrick Stach for ideas on this */

#define MAX_OUTER_THREADS 256

__global__ void
lanczos_kernel_outer_prod(v_t *x, v_t *y,
			v_t *xy, uint32 n) 
{
	uint32 i, w_x, w_y;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 block_id = threadIdx.x;
	__shared__ uint64 scratch[3 * MAX_OUTER_THREADS];

	for (w_x = 0; w_x < VWORDS; w_x++) {
		for (w_y = 0; w_y < VWORDS; w_y++) {
			uint64 *s = scratch + (block_id & ~0x1f);
			scratch[block_id + 0*MAX_OUTER_THREADS] = 0;
			scratch[block_id + 1*MAX_OUTER_THREADS] = 0;
			scratch[block_id + 2*MAX_OUTER_THREADS] = 0;

			for (i = grid_id; i < n; i += num_threads) {
				uint32 j;
				uint32 k = block_id & 0x1f;
				uint64 xi = x[i].w[w_x];
				uint64 yi = y[i].w[w_y];

				if (k != 0)
					xi = (xi >> (2 * k)) | (xi << (64 - (2 * k)));

#pragma unroll
				for (j = 0; j < 32; j++) {
					uint32 off = bfe(xi, 2 * j, 2);
					uint64 tmp = yi;

					if (off == 0) {
						tmp = 0;
						off = 1;
					}

					s[((k + j) & 0x1f) + 
						MAX_OUTER_THREADS * (off - 1)] ^= tmp;
				}
			}

			s = scratch + block_id;
			__syncthreads();
			s[0*MAX_OUTER_THREADS] ^= s[2*MAX_OUTER_THREADS];
			s[1*MAX_OUTER_THREADS] ^= s[2*MAX_OUTER_THREADS];
			__syncthreads();

			for (i = MAX_OUTER_THREADS / 2; i >= 32; i >>= 1) {
				if (block_id < i) {
					s[0*MAX_OUTER_THREADS] ^= s[0*MAX_OUTER_THREADS + i];
					s[1*MAX_OUTER_THREADS] ^= s[1*MAX_OUTER_THREADS + i];
				}
				__syncthreads();
			}

			if (block_id < 32) {
				uint64 res = scratch[block_id];
				i = 2 * block_id;
				atomicXor(&xy[64 * w_x + i].w[w_y], res);
			}
			else if (block_id < 64) {
				uint64 res = scratch[MAX_OUTER_THREADS + block_id - 32];
				i = 2 * block_id - 64 + 1;
				atomicXor(&xy[64 * w_x + i].w[w_y], res);
			}
			__syncthreads();
		}
	}
}

#ifdef __cplusplus
}
#endif
